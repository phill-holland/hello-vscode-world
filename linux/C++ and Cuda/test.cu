#include "hip/hip_runtime.h"
#include "test.cuh"

__global__ void TestCopy(int4 *source, int4 *destination, const int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
        destination[idx] = source[idx];
    }
}

void cuda::test::init()
{
    int length = 256;

    if (hipMalloc((void**)&cudaInt4ArrayA, sizeof(int4) * length) != ::hipSuccess) return;
    if (hipMalloc((void**)&cudaInt4ArrayB, sizeof(int4) * length) != ::hipSuccess) return;
}

void cuda::test::run()
{
    int grids = 2;
    int threads = 128;

    TestCopy<<<grids, threads>>>(cudaInt4ArrayA, cudaInt4ArrayB, 256);
}

void cuda::test::cleanup()
{
    hipFree(cudaInt4ArrayB);
    hipFree(cudaInt4ArrayA);
}